#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h> 
#include <hip/hip_runtime.h> 
#include <f2c.h>  
#include <clapack.h> 
#include <cstdlib>
#include <ctime>

//nvcc Open.cu -lm -lcuda -lcudart -llapack -lblas -lf2c -lcublas -arch=sm_20 -o open
//I.V. Ionova, E.A. Carter, "Error vector choice in direct inversion in the iterative subspace method, J. Compt. Chem. 17, 1836-1847, 1996. 

double *BlasResult;
hipblasStatus_t stat; 
hipblasHandle_t handle;
int n_mol, n_seg;
int *N;
double *chi, *GN, *phibulk, *theta, *n;
int block_size=256,i,j,k,kk,k_diis,kk_diis,m,mm, s,Mx,My,Mz,M,jx,jy,it,bx1,by1,bz1,bxm,bym,bzm,iv,iiv;
double sigma, error = 1, tolerance = 1e-7, eta = 0.1, normC, pi=4.0*atan(1.0);
<<<<<<< .mine
double *Aij,*Ci,*Apij,*H_phi,*H_mask,*H_u,*mask,*phi,*phi_pinned,*phitot,*G1,*alpha,*Gg_f,*Gg_b,*phi_side,*x,*x0,*g,*xR,*x_x0;
double *u,*f, *r;
=======
double *Aij,*Ci,*Apij,*H_phi,*H_mask,*H_u,*mask,*phi,*phi_pinned,*phitot,*G1,*alpha,*Gg_f,*Gg_b,*phi_side,*x,*x0,*g,*xR,*x_x0;
double *u,*f,*r,*r0;
double *AAij,*CCi,*AApij,*r_r0;
>>>>>>> .r399

__global__ void times(double *P, double *A, double *B, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]=A[idx]*B[idx];
}
__global__ void addtimes(double *P, double *A, double *B, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]+=A[idx]*B[idx];
}
__global__ void norm(double *P, double C, int M){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] *= C;
}
__global__ void zero(double *P, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] = 0.0;
}
__global__ void cp (double *P, double *A, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] = A[idx];
}
__global__ void yisaminb(double *Y, double *A,double *B, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) Y[idx] = A[idx]-B[idx];
}
__global__ void yplusisctimesx(double *Y, double *X, double C, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) Y[idx] += C*X[idx];
}
__global__ void add(double *P, double *A, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]+=A[idx];
}
__global__ void dubble(double *P, double *A, double norm, int M){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]*=norm/A[idx];
}
__global__ void boltzmann(double *P, double *A, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]=exp(-A[idx]);
}
__global__ void putalpha(double *g,double *phitot,double *phi_side,double chi,double phibulk,int M){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) g[idx] = g[idx] - chi*(phi_side[idx]/phitot[idx]-phibulk);
}
__global__ void addg(double *g, double *phitot, double *alpha, int M) { 
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) {
		g[idx]= g[idx] -alpha[idx] +1/phitot[idx]-1;
	}
}
__global__ void bx(double *P, int mmx, int My, int Mz, int bx1, int bxm, int jx, int jy){
	int idx, jx_mmx=jx*mmx, jx_bxm=jx*bxm, bx1_jx=bx1*jx;
	int yi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (yi<My && zi<Mz) {
		idx=jy*yi+zi;
		P[idx]=P[bx1_jx+idx];
		P[jx_mmx+idx]=P[jx_bxm+idx];
	}
}
__global__ void b_x(double *P, int mmx, int My, int Mz, int bx1, int bxm, int jx, int jy){
	int idx, jx_mmx=jx*mmx;
	int yi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (yi<My && zi<Mz) {
		idx=jy*yi+zi;
		P[idx]=0;
		P[jx_mmx+idx]=0;
	}
}
__global__ void by(double *P, int Mx, int mmy, int Mz, int by1, int bym, int jx, int jy){
	int idx, jy_mmy=jy*mmy, jy_bym=jy*bym, jy_by1=jy*by1;
	int xi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && zi<Mz) {
		idx=jx*xi+zi;
		P[idx]=P[jy_by1+idx];
		P[jy_mmy+idx]=P[jy_bym+idx];
	}
}
__global__ void b_y(double *P, int Mx, int mmy, int Mz, int by1, int bym, int jx, int jy){
	int idx, jy_mmy=jy*mmy;
	int xi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && zi<Mz) {
		idx=jx*xi+zi;
		P[idx]=0;
		P[jy_mmy+idx]=0;
	}
}
__global__ void bz(double *P, int Mx, int My, int mmz, int bz1, int bzm, int jx, int jy){
	int idx, xi =blockIdx.x*blockDim.x+threadIdx.x, yi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && yi<My) {
		idx=jx*xi+jy*yi;
		P[idx]=P[idx+bz1];
		P[idx+mmz]=P[idx+bzm];
	}
}
__global__ void b_z(double *P, int Mx, int My, int mmz, int bz1, int bzm, int jx, int jy){
	int idx, xi =blockIdx.x*blockDim.x+threadIdx.x, yi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && yi<My) {
		idx=jx*xi+jy*yi;
		P[idx]=0; 
		P[idx+mmz]=0;
	}
}
double Dot(double *x,double *y,int M){ 
	double result;
 	hipblasDdot(handle,M,x,1,y,1,BlasResult);
	hipMemcpy(&result,BlasResult,sizeof(double),hipMemcpyDeviceToHost);
	return result;
}
double Sum(double *x,int M){
	double result;
 	hipblasDasum(handle,M,x,1,BlasResult);
	hipMemcpy(&result,BlasResult,sizeof(double),hipMemcpyDeviceToHost);
	return result;
}
bool GPU_present() {
    int deviceCount =0; hipGetDeviceCount(&deviceCount);
    if (deviceCount ==0) printf("There is no device supporting Cuda.\n");
    else hipDeviceReset();
	return deviceCount > 0;
}
double *AllOnDev(int N) {
	double *X;
	if (hipSuccess != hipMalloc((void **) &X, sizeof(double)*N))
	printf("Memory allocation on GPU failed.\n Please reduce size of lattice and/or chain length(s)\n");
	return X;
}
void TransferDataToHost(double *H, double *D) {
	hipMemcpy(H, D, sizeof(double)*M,hipMemcpyDeviceToHost);
}
void TransferDataToDevice(int M, double *H, double *D ) { 
	hipMemcpy(D, H, sizeof(double)*M,hipMemcpyHostToDevice);
}
void AddTimes(double *P, double *A, double *B, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	addtimes<<<n_blocks,block_size>>>(P,A,B,M);
}
void Times(double *P, double *A, double *B, int M){ int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	times<<<n_blocks,block_size>>>(P,A,B,M);
}
void Norm(double *P, double C, int M){
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	norm<<<n_blocks,block_size>>>(P,C,M);
}
void Zero(double* P, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	zero<<<n_blocks,block_size>>>(P,M);
}
void Cp(double *P,double *A, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	cp<<<n_blocks,block_size>>>(P,A,M);
}
void YisAminB(double *Y, double *A, double *B, int M){ int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	yisaminb<<<n_blocks,block_size>>>(Y,A,B,M);
}
void Add(double *P, double *A, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	add<<<n_blocks,block_size>>>(P,A,M);
}
void Dubble(double *P, double *A, double norm){ 
       int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	dubble<<<n_blocks,block_size>>>(P,A,norm,M);
}
void Boltzmann(double *P, double *A){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	boltzmann<<<n_blocks,block_size>>>(P,A,M);
}
void PutAlpha(double *g, double *phitot, double *phi_side, double chi, double phibulk){
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	putalpha<<<n_blocks,block_size>>>(g,phitot,phi_side,chi,phibulk,M);
}
void SetBoundaries(double *P) {
	dim3 dimBlock(16,16);
	dim3 dimGridz((Mx+dimBlock.x+1)/dimBlock.x,(My+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridy((Mx+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridx((My+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	bx<<<dimGridx,dimBlock>>>(P,Mx+1,My+2,Mz+2,bx1,bxm,jx,jy);
	by<<<dimGridy,dimBlock>>>(P,Mx+2,My+1,Mz+2,by1,bym,jx,jy);
	bz<<<dimGridz,dimBlock>>>(P,Mx+2,My+2,Mz+1,bz1,bzm,jx,jy);
}
void RemoveBoundaries(double *g) {
	dim3 dimBlock(16,16);
	dim3 dimGridz((Mx+dimBlock.x+1)/dimBlock.x,(My+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridy((Mx+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridx((My+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	b_x<<<dimGridx,dimBlock>>>(g,Mx+1,My+2,Mz+2,bx1,bxm,jx,jy);
	b_y<<<dimGridy,dimBlock>>>(g,Mx+2,My+1,Mz+2,by1,bym,jx,jy);
	b_z<<<dimGridz,dimBlock>>>(g,Mx+2,My+2,Mz+1,bz1,bzm,jx,jy);
}
void Side(double *phi_side, double *phi) {
	Zero(phi_side,M); SetBoundaries(phi);
	Add(phi_side+jx,phi,M-jx); Add(phi_side,phi+jx,M-jx);
	Add(phi_side+jy,phi,M-jy); Add(phi_side,phi+jy,M-jy);
	Add(phi_side+1, phi,M-1);  Add(phi_side,phi+1, M-1);
	Norm(phi_side,1.0/6.0,M);
}
void Propagate(double *G, double *G1, int s_from, int s_to) {
	double *gs = G+M*(s_to-1), *gs_1 = G+M*(s_from-1), *g = G1;
	SetBoundaries(gs_1);
	Times(gs+jx,gs_1,g+jx,M-jx); AddTimes(gs,gs_1+jx,g,M-jx);
	AddTimes(gs+jy,gs_1,g+jy,M-jy); AddTimes(gs,gs_1+jy,g,M-jy);
    AddTimes(gs+1, gs_1,g+1, M-1);  AddTimes(gs,gs_1+1, g,M-1);
   	Norm(gs,1.0/6.0,M);
};
void ComputePhi(){
	int N1=N[1];
	int Ndiv2=N1/2;
	for (int i=0; i<n_seg; i++) { Boltzmann(G1+i*M,u+i*M); SetBoundaries(G1+i*M); }
	Cp(phi,G1,M);//phi for solvent;	
	Cp(Gg_f,G1+M,M); for (s=2; s<=Ndiv2; s++) Propagate(Gg_f,G1+M,s-1,s); 
	Propagate(Gg_f,G1+2*M,Ndiv2,Ndiv2+1);
	Times(Gg_f+Ndiv2*M,Gg_f+Ndiv2*M,mask,M);
	for (int s=Ndiv2+2; s<=N1; s++) Propagate(Gg_f,G1+2*M,s-1,s); 
	Cp(Gg_b+(N1%2)*M,G1+2*M,M); 
	Times(phi+2*M,Gg_f+(N1-1)*M,Gg_b+(N1%2)*M,M);	
	for (int s=N1-1; s>Ndiv2; s--) {
		Propagate(Gg_b,G1+2*M,((s+1)%2)+1,(s%2)+1); 
		AddTimes(phi+2*M,Gg_f+(s-1)*M,Gg_b+(s%2)*M,M);
	}
	Times(phi_pinned,Gg_f+Ndiv2*M,Gg_b+((Ndiv2+1)%2)*M,M); 
	Times(Gg_b+((Ndiv2+1)%2)*M,Gg_b+((Ndiv2+1)%2)*M,mask,M);
	Zero(phi+M,M);
	for (int s=Ndiv2; s>=1; s--) {
		Propagate(Gg_b,G1+M,((s+1)%2)+1,(s%2)+1); 
		AddTimes(phi+M,Gg_f+(s-1)*M,Gg_b+(s%2)*M,M); 
	}
	RemoveBoundaries(Gg_f+(N1-1)*M);
	GN[1]=Sum(Gg_f+(N1-1)*M,M); 
	for (int i=1; i<n_seg; i++) Dubble(phi+i*M,G1+i*M,n[1]/GN[1]); 
	Dubble(phi_pinned,G1+2*M,n[1]/GN[1]);
	phibulk[0]=1; phibulk[1]=phibulk[2]=0;
}
void AddG(double *g, double *phitot, double *alpha){
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	addg<<<n_blocks,block_size>>>(g,phitot,alpha,M);
}
void ComputeG(){
	bool doit; 
	ComputePhi();
	Cp(phitot,phi,M); for (int i=1; i<n_seg; i++) Add(phitot,phi+i*M,M);
	for (int i=0; i<n_seg; i++) { 
		doit = false; 
		for (int j=0; j<n_seg; j++) {
			if (chi[i+n_seg*j] !=0) doit=true;
		}
		if (doit) Side(phi_side+i*M,phi+i*M); //only needed when the respective chi calls for it. 
	}
	Cp(g,u,iv); 
	for (int i=0; i<n_seg; i++) for (int j=0; j<n_seg; j++) {
		if ((i != j) && (chi[i+n_seg*j] !=0) ) {
			PutAlpha(g+i*M,phitot,phi_side+j*M,chi[i+n_seg*j],phibulk[j]);
		}
	} //now g contains segment type dependent alpha.
	Cp(alpha,g,M); for (int i=1; i<n_seg; i++) Add(alpha,g+i*M,M);  Norm(alpha,1.0/n_seg,M); //alpha contains average alpha
	for (int i=0; i<n_seg; i++) {AddG(g+i*M,phitot,alpha); RemoveBoundaries(g+i*M);}
}
void YplusisCtimesX(double *Y, double *X, double C, int M) {
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	yplusisctimesx<<<n_blocks,block_size>>>(Y,X,C,M);
}
void Ax(double* A, double* X, int N){//From Ax_B; below B is not used: it is assumed to contain a row of unities.
	double *U = new double[N*N];
	double *S = new double[N];
	double *VT = new double[N*N];
	integer MM = (integer)N, NN = (integer)N;
	integer LDA=MM, LDU=MM, LDVT=NN, INFO, LWORK;
	int lwork;
	double WKOPT;
	double* WORK;
	char JOBU='S'; //'S' is nodig om alleen de eerste N colommen in U te schrijven.
	char JOBVT='A';

	LWORK = -1; //grootte hulpgeheugen aanvragen
	dgesvd_( &JOBU, &JOBVT, &MM, &NN, A, &LDA, S, U, &LDU, VT, &LDVT, &WKOPT, &LWORK, &INFO );
	lwork = (int)WKOPT;
	WORK = (double*)malloc( lwork*sizeof(double) );
	LWORK = (integer)lwork; //nu uitrekenen.
	dgesvd_( &JOBU, &JOBVT, &MM, &NN, A, &LDA, S, U, &LDU, VT, &LDVT, WORK, &LWORK, &INFO );
	if (INFO >0) { //error message genereren
	};
	delete WORK;
	for (int i=0; i<N; i++) X[i]=0;
	for (int i=0; i<N; i++) for (int j=0; j<N; j++) X[i] += U[i*N + j];//*B[j];
	for (int i=0; i<N; i++) {S[i] = X[i]/S[i]; X[i]=0;} //S is use decause it is no longer needed.
	for (int i=0; i<N; i++) for (int j=0; j<N; j++) X[i] += VT[i*N + j]*S[j];
	delete U,S,VT;
}
void DIIS(double *x, double *x_x0, double *xR, double *Aij, double *Apij, double *Ci, int k, int m, int iv) {
	double normC=0; int posi;  
	if (k_diis>m) { k_diis =m;
		for (int i=1; i<m; i++) for (int j=1; j<m; j++) 
		Aij[m*(i-1)+j-1]=Aij[m*i+j]; //remove oldest elements
	}
	for (int i=0; i<k_diis; i++) {
		posi = k-k_diis+1+i; if (posi<0) posi +=m; 
		Aij[i+m*(k_diis-1)] = Aij[k_diis-1+m*i] = Dot(x_x0+posi*iv, x_x0+k*iv,iv);	
	}
	for (int i=0; i<k_diis; i++) for (int j=0; j<k_diis; j++) {
		Apij[j+k_diis*i] = Aij[j+m*i];
	}
	Ax(Apij,Ci,k_diis);		
	for (int i=0; i<k_diis; i++) normC +=Ci[i];
	for (int i=0; i<k_diis; i++) Ci[i] =Ci[i]/normC; 
	Zero(x,iv);
	posi = k-k_diis+1; if (posi<0) posi +=m; 
		
	YplusisCtimesX(x,xR+posi*iv,Ci[0],iv); //pv = Ci[0]*xR[0];
	for (int i=1; i<k_diis; i++) { 
		posi = k-k_diis+1+i; if (posi<0) posi +=m; 
		YplusisCtimesX(x,xR+posi*iv,Ci[i],iv); 
	}
}
double Helmholtz(){
	double F_Helmholtz=0;
	F_Helmholtz = -n[1]*log(GN[1]/n[1]/N[1]);
	RemoveBoundaries(alpha);
	F_Helmholtz -= Sum(alpha,M);
	return F_Helmholtz;
}
double SCF() {
	TransferDataToDevice(M, H_mask, mask); 
	Zero(x,iv); Zero(x0,iv);
	it=0; k_diis=1; k=0;
	ComputeG();
	YplusisCtimesX(x,g,-eta,iv);
	YisAminB(x_x0,x,x0,iv);
	Cp(xR,x,iv); 
	error = sqrt(Dot(g,g,iv));
	printf("DIIS has been notified\n");
	printf("Your guess = %1e \n",error);
	while (error > tolerance && it < 1000) {
		it++;
		Cp(x0,x,iv); ComputeG();
		k=it % m; k_diis++; //plek voor laatste opslag
		YplusisCtimesX(x,g,-eta,iv);
		Cp(xR+k*iv,x,iv); YisAminB(x_x0+k*iv,x,x0,iv); 	
		DIIS(x,x_x0,xR,Aij,Apij,Ci,k,m,iv); 
		error = sqrt(Dot(g,g,iv));
		printf("it = %i error = %1e \n",it,error);
	}
	return Helmholtz();
}
double random_d() {
	return rand()/(RAND_MAX+1.0);
}
int random_int(int low, int high) {
	int range = high-low+1;
	return low+int(range*random_d());
}
double random(double low, double high) {
	double range = high-low;
	return low+range*random_d();
}
bool Stamp() {
	double rx,ry,rz; int pos;
	double rx1,ry1,rz1;
	int xr,yr,zr,xr1,yr1,zr1,x,y,z;
	//double volume;
	Zero(mask,M); TransferDataToHost(H_mask,mask);
	for (int i = 0; i<n[1]; i++) { 
		//volume=0;
		rx=r[3*i]; ry=r[3*i+1]; rz=r[3*i+2];
		rx1=rx+1; ry1=ry+1; rz1=rz+1;
		printf("(x,y,z) %1f %1f %1f \n",rx,ry,rz);
		xr=rx; yr=ry; zr=rz; //afgerond coordinaten
		xr1=rx1; yr1=ry1; zr1=rz1;
		for (int i=xr; i<=xr1; i++) for (int j=yr; j<=yr1; j++) for (int k=zr; k<=zr1; k++) {	
			//if (i<rx) dx=xr+1-rx; else if (i>rx+1) dx=xr1-i; else if (i+1<rx) dx=1.0; else dx=rx1-i; 
			//if (j<ry) dy=yr+1-ry; else if (j>ry+1) dy=yr1-j; else if (j+1<ry) dy=1.0; else dy=ry1-j;
			//if (k<rz) dz=zr+1-rz; else if (k>rz+1) dz=zr1-k; else if (k+1<rz) dz=1.0; else dz=rz1-k;
			x=i%Mx; y=j%My; z=k%Mz; if (x==0) x=Mx; if (y==0) y=My; if (z==0) z=Mz;			
			pos = jx*x+jy*y+z; if (H_mask[pos]>0.0) {printf("Core overlap \n"); return false;}
			H_mask[pos]=1; 
			//volume +=H_mask[pos];
			//if (H_mask[pos]>0) printf("H(x,y,z) ( %i , %i , %i ) %1f \n",x,y,z,H_mask[pos]);
		}
		//printf("Volume %1f %f\n",volume,pow(2,3));
	}
	return true;
}
void GetForces() {
	double rx,ry,rz; int pos;
	double fx,fy,fz;
	double rx1,ry1,rz1;
	double mean_u;
	int Pos [2][2][2];
	int xr,yr,zr,xr1,yr1,zr1,x,y,z;
	SetBoundaries(u+2*M); TransferDataToHost(H_u,u+2*M);
	SetBoundaries(phi_pinned); TransferDataToHost(H_phi,phi_pinned);
	for (int p=0; p<n[1]; p++) {
		rx=r[3*p]; ry=r[3*p+1]; rz=r[3*p+2];
		rx1=rx+1; ry1=ry+1; rz1=rz+1;
		xr=rx; yr=ry; zr=rz; //afgerond coordinaten
		xr1=rx1; yr1=ry1; zr1=rz1;
		fx=fy=fz=0;
		for (int i=xr; i<=xr1; i++) for (int j=yr; j<=yr1; j++) for (int k=zr; k<=zr1; k++) {
			x=i%Mx; y=j%My; z=k%Mz; if (x==0) x=Mx; if (y==0) y=My; if (z==0) z=Mz;	
			pos = jx*x+jy*y+z;
			//printf("%i %i %i %1f %1f %1f \n",x,y,z,H_mask[pos], H_u[pos], H_phi[pos]);
			if (i==xr) fx +=H_u[pos];
			if (j==yr) fy +=H_u[pos];
			if (k==zr) fz +=H_u[pos];
			if (i==xr1) fx -=H_u[pos];
			if (j==yr1) fy -=H_u[pos];
			if (k==zr1) fz -=H_u[pos];
		}
		 
		f[3*p] = fx/4; f[3*p+1] = fy/4; f[3*p+2] = fz/4;
		printf("i: fx,fy,fz: %i %1f %1f %1f \n",p,f[3*p],f[3*p+1],f[3*p+2]);
	
		fx=fy=fz=0; mean_u=0;
		for (int i=xr; i<=xr1; i++) for (int j=yr; j<=yr1; j++) for (int k=zr; k<=zr1; k++) {
			x=i%Mx; y=j%My; z=k%Mz; if (x==0) x=Mx; if (y==0) y=My; if (z==0) z=Mz;	
			pos = jx*x+jy*y+z;
			
			//printf("%i %i %i %1f %1f %1f \n",x,y,z,H_mask[pos], H_u[pos], H_phi[pos]);
			if (i==xr) fx +=H_phi[pos];
			if (j==yr) fy +=H_phi[pos];
			if (k==zr) fz +=H_phi[pos];
			if (i==xr1) fx -=H_phi[pos];
			if (j==yr1) fy -=H_phi[pos];
			if (k==zr1) fz -=H_phi[pos];
			mean_u +=H_u[pos];
		}
		mean_u /=8;
		f[3*p] = mean_u*fx; f[3*p+1] = mean_u*fy; f[3*p+2] = mean_u*fz;
		printf("i: fx,fy,fz: %i %1f %1f %1f \n",p, f[3*p],f[3*p+1],f[3*p+2]);

		fx=fy=fz=0; 
		for (int i=xr; i<=xr1; i++) for (int j=yr; j<=yr1; j++) for (int k=zr; k<=zr1; k++) {
			x=i%Mx; y=j%My; z=k%Mz; if (x==0) x=Mx; if (y==0) y=My; if (z==0) z=Mz;	
			pos = jx*x+jy*y+z;
			Pos[i%xr][j%yr][k%zr]=pos;	
		}
		f[3*p] = (H_u[Pos[0][0][0]]-H_u[Pos[1][0][0]])*(H_phi[Pos[0][0][0]]+H_phi[Pos[1][0][0]]); 
		f[3*p] += (H_u[Pos[0][0][1]]-H_u[Pos[1][0][1]])*(H_phi[Pos[0][0][1]]+H_phi[Pos[1][0][1]]);
		f[3*p] += (H_u[Pos[0][1][0]]-H_u[Pos[1][1][0]])*(H_phi[Pos[0][1][0]]+H_phi[Pos[1][1][0]]);
		f[3*p] += (H_u[Pos[0][1][1]]-H_u[Pos[1][1][1]])*(H_phi[Pos[0][1][1]]+H_phi[Pos[1][1][1]]);

		f[3*p+1] = (H_u[Pos[0][0][0]]-H_u[Pos[0][1][0]])*(H_phi[Pos[0][0][0]]+H_phi[Pos[0][1][0]]); 
		f[3*p+1] += (H_u[Pos[1][0][0]]-H_u[Pos[1][1][0]])*(H_phi[Pos[1][0][0]]+H_phi[Pos[1][1][0]]);
		f[3*p+1] += (H_u[Pos[0][0][1]]-H_u[Pos[0][1][1]])*(H_phi[Pos[0][0][1]]+H_phi[Pos[0][1][1]]);
		f[3*p+1] += (H_u[Pos[1][0][1]]-H_u[Pos[1][1][1]])*(H_phi[Pos[1][0][1]]+H_phi[Pos[1][1][1]]);

		f[3*p+2] = (H_u[Pos[0][0][0]]-H_u[Pos[0][0][1]])*(H_phi[Pos[0][0][0]]+H_phi[Pos[0][0][1]]);
		f[3*p+2] += (H_u[Pos[1][0][0]]-H_u[Pos[1][0][1]])*(H_phi[Pos[1][0][0]]+H_phi[Pos[1][0][1]]);
		f[3*p+2] += (H_u[Pos[0][1][0]]-H_u[Pos[0][1][1]])*(H_phi[Pos[0][1][0]]+H_phi[Pos[0][1][1]]);
		f[3*p+2] += (H_u[Pos[1][1][0]]-H_u[Pos[1][1][1]])*(H_phi[Pos[1][1][0]]+H_phi[Pos[1][1][1]]);

		printf("i: fx,fy,fz: %i %1f %1f %1f \n",p,f[3*p],f[3*p+1],f[3*p+2]);

		f[3*p] = (H_phi[Pos[0][0][0]]-H_phi[Pos[1][0][0]])*(H_u[Pos[0][0][0]]+H_u[Pos[1][0][0]])/2; 
		f[3*p] += (H_phi[Pos[0][0][1]]-H_phi[Pos[1][0][1]])*(H_u[Pos[0][0][1]]+H_u[Pos[1][0][1]])/2;
		f[3*p] += (H_phi[Pos[0][1][0]]-H_phi[Pos[1][1][0]])*(H_u[Pos[0][1][0]]+H_u[Pos[1][1][0]])/2;
		f[3*p] += (H_phi[Pos[0][1][1]]-H_phi[Pos[1][1][1]])*(H_u[Pos[0][1][1]]+H_u[Pos[1][1][1]])/2;

		f[3*p+1] = (H_phi[Pos[0][0][0]]-H_phi[Pos[0][1][0]])*(H_u[Pos[0][0][0]]+H_u[Pos[0][1][0]])/2; 
		f[3*p+1] += (H_phi[Pos[1][0][0]]-H_phi[Pos[1][1][0]])*(H_u[Pos[1][0][0]]+H_u[Pos[1][1][0]])/2;
		f[3*p+1] += (H_phi[Pos[0][0][1]]-H_phi[Pos[0][1][1]])*(H_u[Pos[0][0][1]]+H_u[Pos[0][1][1]])/2;
		f[3*p+1] += (H_phi[Pos[1][0][1]]-H_phi[Pos[1][1][1]])*(H_u[Pos[1][0][1]]+H_u[Pos[1][1][1]])/2;

		f[3*p+2] = (H_phi[Pos[0][0][0]]-H_phi[Pos[0][0][1]])*(H_u[Pos[0][0][0]]+H_u[Pos[0][0][1]])/2;
		f[3*p+2] += (H_phi[Pos[1][0][0]]-H_phi[Pos[1][0][1]])*(H_u[Pos[1][0][0]]+H_u[Pos[1][0][1]])/2;
		f[3*p+2] += (H_phi[Pos[0][1][0]]-H_phi[Pos[0][1][1]])*(H_u[Pos[0][1][0]]+H_u[Pos[0][1][1]])/2;
		f[3*p+2] += (H_phi[Pos[1][1][0]]-H_phi[Pos[1][1][1]])*(H_u[Pos[1][1][0]]+H_u[Pos[1][1][1]])/2;

		printf("i: fx,fy,fz: %i %1f %1f %1f \n",p,f[3*p],f[3*p+1],f[3*p+2]);

	}
}

main() {  
	hipDeviceReset();

	stat = hipblasCreate(&handle); if (stat !=HIPBLAS_STATUS_SUCCESS) {printf("CUBLAS failed \n");}
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	printf("n_layers x (10): "); scanf("%d", &Mx);
	printf("n_layers y (10): "); scanf("%d", &My);
	printf("n_layers z (10): "); scanf("%d", &Mz);	
	n_mol=2;	
	n = new double[n_mol]; n[0]=0; 
	phibulk = new double[n_mol]; 
	GN = new double[n_mol];
	N = new int[n_mol];
	chi = new double[n_seg*n_seg];
	n_seg=3;
	printf("mol 1 : solvent S \n"); N[0]=1;
	printf("mol 2 : (A)N-B'-(B)N: Give N:"); scanf("%d", &N[1]); 
	N[1]=2*N[1]+1;
	printf("give n mol 2: "); scanf("%lf", &n[1]);
	
	int seg1, seg2;
	//printf("chi(S,A) "); scanf("%lf", &CHI); 
	seg1=0; seg2=1; chi[seg1+n_seg*seg2]=chi[seg2+n_seg*seg1]=0;
	//printf("chi(S,B) "); scanf("%lf", &CHI); 
	seg1=0; seg2=2; chi[seg1+n_seg*seg2]=chi[seg2+n_seg*seg1]=0;
	//printf("chi(A,B) "); scanf("%lf", &CHI); 
	seg1=1; seg2=2; chi[seg1+n_seg*seg2]=chi[seg2+n_seg*seg1]=0;
	
//	printf("Tolerance (1e-7): "); scanf("%lf", &tolerance);
// 	printf("Regularisation parameter, eta (0.1): "); scanf("%lf", &eta);
//	printf("Memory depth (m) : "); scanf("%d", &m);
m=10;
	jx = (My+2)*(Mz+2); jy = Mz+2; M=(Mx+2)*(My+2)*(Mz+2);
	bx1=Mx; bxm=1; by1=My; bym=1; bz1=Mz; bzm=1; // periodic
	iv=n_seg*M;

	Aij = new double[m*m]; for (int i=0; i<m*m; i++) Aij[i]=0;
	Ci = new double[m]; for (int i=0; i<m; i++) Ci[i]=0;
	Apij = new double[m*m]; for (int i=0; i<m*m; i++) Apij[i]=0;

	H_phi = new double[M]; 	H_mask = new double[M]; H_u = new double[M]; mask = (double*)AllOnDev(M); phi = (double*)AllOnDev(iv);
	phi_pinned = (double*)AllOnDev(M); phitot = (double*)AllOnDev(M); G1 = (double*)AllOnDev(iv); alpha = (double*)AllOnDev(M);
	Gg_f = (double*)AllOnDev(M*N[1]); Gg_b = (double*)AllOnDev(M*2); phi_side = (double*)AllOnDev(iv); x = (double*)AllOnDev(iv);
	x0 = (double*)AllOnDev(iv); g = (double*)AllOnDev(iv); xR = (double*)AllOnDev(m*iv); x_x0 = (double*)AllOnDev(m*iv);
	BlasResult = (double*)AllOnDev(1);
	r = new double[3*N[1]];
	f = new double[3*N[1]]; 

	u = x; 
	//srand(time(0));
	bool succes=false;
	while (!succes) {
		int j=0;
		for (int i=0; i<n[1]; i++) {r[j++]=random(1.0,1.0*Mx); r[j++]=random(1.0,1.0*My); r[j++]=random(1.0,1.0*Mz);}
		succes = Stamp();
	}

	double Free_energy = SCF();
	GetForces();
	

	
	//TransferDataToHost(H_phi,phi);
	//printf("z \t phi  \n"); for (int zz=1; zz<=Mz; zz++) printf(" %i \t %1f \n", zz, H_phi[jx+jy+zz]);
	//TransferDataToHost(H_phi,phi+1*M);
	//printf("z \t phi  \n"); for (int zz=1; zz<=Mz; zz++) printf(" %i \t %1f \n", zz, H_phi[jx+jy+zz]);
	//TransferDataToHost(H_phi,phi+2*M);
	//printf("z \t phi  \n"); for (int zz=1; zz<=Mz; zz++) printf(" %i \t %1f \n", zz, H_phi[jx+jy+zz]);

	free(H_phi);
	hipFree(phi); hipFree(x);
	hipFree(G1);hipFree(alpha);hipFree(Gg_f);hipFree(Gg_b);hipFree(phi_side);
	hipFree(x_x0);
	hipblasDestroy(handle);
	return(0);
};
