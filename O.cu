#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h> //versie 4?   
#include <hip/hip_runtime.h> 
#include <f2c.h>  
#include <clapack.h> 

//nvcc OpenBoxLBFGSDIIS.cu -lm -lcuda -lcudart -llapack -lblas -lf2c -lcublas -arch=sm_20 -o open
//I.V. Ionova, E.A. Carter, "Error vector choice in direct inversion in the iterative subspace method, J. Compt. Chem. 17, 1836-1847, 1996. 

double *DotResult;
hipblasStatus_t stat; 
hipblasHandle_t handle;
int block_size=256,i,j,k,k_diis,m,s,Mx,My,Mz,N_g,M,jx,jy,it,bx1,by1,bz1,bxm,bym,bzm,pol2,pol,sol,iv;
double sigma, chi, error = 1, tolerance = 1e-7, eta = 0.1, normC, GN, theta;
double *Aij,*Ci,*Apij; 
double *H_phi,*H_mask,*mask,*phi,*phitot,*G1,*alpha,*Gg_f,*Gg_b,*phi_side,*x,*x0,*g,*xR,*x_x0;
double *u;
__global__ void times(double *P, double *A, double *B, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]=A[idx]*B[idx];
}
__global__ void addtimes(double *P, double *A, double *B, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]+=A[idx]*B[idx];
}
__global__ void norm(double *P, double C, int M){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] *= C;
}
__global__ void zero(double *P, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] = 0.0;
}
__global__ void cp (double *P, double *A, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] = A[idx];
}
__global__ void yisaminb(double *Y, double *A,double *B, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) Y[idx] = A[idx]-B[idx];
}
__global__ void yplusisctimesx(double *Y, double *X, double C, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) Y[idx] += C*X[idx];
}
__global__ void add(double *P, double *A, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]+=A[idx];
}
__global__ void dubble(double *P, double *A, double norm, int M){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]*=norm/A[idx];
}
__global__ void boltzmann(double *P, double *A, int M){int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx]=exp(-A[idx]);
}
__global__ void putalpha(double *g,double *u,double *phitot,double *phi_side,double chi,double phibulk,int M){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) g[idx]=u[idx]-chi*(phi_side[idx]/phitot[idx]-phibulk);
}
__global__ void addg(double *g, double *phitot, double *alpha, int M) { 
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) {
		g[idx]=g[idx]-alpha[idx]/2 +1/phitot[idx]-1;
		g[idx+M]=g[idx+M]-alpha[idx]/2 +1/phitot[idx]-1;
	}
}
__global__ void bx(double *P, int mmx, int My, int Mz, int bx1, int bxm, int jx, int jy){
	int idx, jx_mmx=jx*mmx, jx_bxm=jx*bxm, bx1_jx=bx1*jx;
	int yi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (yi<My && zi<Mz) {
		idx=jy*yi+zi;
		P[idx]=P[bx1_jx+idx];
		P[jx_mmx+idx]=P[jx_bxm+idx];
	}
}
__global__ void b_x(double *P, int mmx, int My, int Mz, int bx1, int bxm, int jx, int jy){
	int idx, jx_mmx=jx*mmx;
	int yi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (yi<My && zi<Mz) {
		idx=jy*yi+zi;
		P[idx]=0;
		P[jx_mmx+idx]=0;
	}
}
__global__ void by(double *P, int Mx, int mmy, int Mz, int by1, int bym, int jx, int jy){
	int idx, jy_mmy=jy*mmy, jy_bym=jy*bym, jy_by1=jy*by1;
	int xi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && zi<Mz) {
		idx=jx*xi+zi;
		P[idx]=P[jy_by1+idx];
		P[jy_mmy+idx]=P[jy_bym+idx];
	}
}
__global__ void b_y(double *P, int Mx, int mmy, int Mz, int by1, int bym, int jx, int jy){
	int idx, jy_mmy=jy*mmy;
	int xi =blockIdx.x*blockDim.x+threadIdx.x, zi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && zi<Mz) {
		idx=jx*xi+zi;
		P[idx]=0;
		P[jy_mmy+idx]=0;
	}
}
__global__ void bz(double *P, int Mx, int My, int mmz, int bz1, int bzm, int jx, int jy){
	int idx, xi =blockIdx.x*blockDim.x+threadIdx.x, yi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && yi<My) {
		idx=jx*xi+jy*yi;
		P[idx]=P[idx+bz1];
		P[idx+mmz]=P[idx+bzm];
	}
}
__global__ void b_z(double *P, int Mx, int My, int mmz, int bz1, int bzm, int jx, int jy){
	int idx, xi =blockIdx.x*blockDim.x+threadIdx.x, yi =blockIdx.y*blockDim.y+threadIdx.y;
	if (xi<Mx && yi<My) {
		idx=jx*xi+jy*yi;
		P[idx]=0; 
		P[idx+mmz]=0;
	}
}
__global__ void initialize (double *Gg_b, double *G1, double *mask, int N, int M){
	double *P=Gg_b+(N-1)*M;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx<M) P[idx] = G1[idx]*mask[idx];
}
double Dot(double *x,double *y,int M){ 
	double result;
 	hipblasDdot(handle,M,x,1,y,1,DotResult);
	hipMemcpy(&result,DotResult,sizeof(double),hipMemcpyDeviceToHost);
	return result;
}
bool GPU_present() {
    int deviceCount =0; hipGetDeviceCount(&deviceCount);
    if (deviceCount ==0) printf("There is no device supporting Cuda.\n");
    else hipDeviceReset();
	return deviceCount > 0;
}
double *AllOnDev(int N) {
	double *X;
	if (hipSuccess != hipMalloc((void **) &X, sizeof(double)*N))
	printf("Memory allocation on GPU failed.\n Please reduce size of lattice and/or chain length(s)\n");
	return X;
}
void TransferDataToHost(double *H, double *D) {
	hipMemcpy(H, D, sizeof(double)*M,hipMemcpyDeviceToHost);
}
void TransferDataToDevice(int M, double *H, double *D ) { 
	hipMemcpy(D, H, sizeof(double)*M,hipMemcpyHostToDevice);
}
void AddTimes(double *P, double *A, double *B, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	addtimes<<<n_blocks,block_size>>>(P,A,B,M);
}
void Times(double *P, double *A, double *B, int M){ int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	times<<<n_blocks,block_size>>>(P,A,B,M);
}
void Norm(double *P, double C, int M){
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	norm<<<n_blocks,block_size>>>(P,C,M);
}
void Zero(double* P, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	zero<<<n_blocks,block_size>>>(P,M);
}
void Cp(double *P,double *A, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	cp<<<n_blocks,block_size>>>(P,A,M);
}
void YisAminB(double *Y, double *A, double *B, int M){ int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	yisaminb<<<n_blocks,block_size>>>(Y,A,B,M);
}
void Add(double *P, double *A, int M){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	add<<<n_blocks,block_size>>>(P,A,M);
}
void Dubble(double *P, double *A, double norm){ 
       int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	dubble<<<n_blocks,block_size>>>(P,A,norm,M);
}
void Boltzmann(double *P, double *A){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	boltzmann<<<n_blocks,block_size>>>(P,A,M);
}
void PutAlpha(double *g, double *u, double *phitot, double *phi_side, double chi, double phibulk){
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	putalpha<<<n_blocks,block_size>>>(g,u,phitot,phi_side,chi,phibulk,M);
}
void SetBoundaries(double *P) {
	dim3 dimBlock(16,16);
	dim3 dimGridz((Mx+dimBlock.x+1)/dimBlock.x,(My+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridy((Mx+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridx((My+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	bx<<<dimGridx,dimBlock>>>(P,Mx+1,My+2,Mz+2,bx1,bxm,jx,jy);
	by<<<dimGridy,dimBlock>>>(P,Mx+2,My+1,Mz+2,by1,bym,jx,jy);
	bz<<<dimGridz,dimBlock>>>(P,Mx+2,My+2,Mz+1,bz1,bzm,jx,jy);
}
void RemoveBoundaries(double *g) {
	dim3 dimBlock(16,16);
	dim3 dimGridz((Mx+dimBlock.x+1)/dimBlock.x,(My+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridy((Mx+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	dim3 dimGridx((My+dimBlock.x+1)/dimBlock.x,(Mz+dimBlock.y+1)/dimBlock.y);
	b_x<<<dimGridx,dimBlock>>>(g,Mx+1,My+2,Mz+2,bx1,bxm,jx,jy);
	b_y<<<dimGridy,dimBlock>>>(g,Mx+2,My+1,Mz+2,by1,bym,jx,jy);
	b_z<<<dimGridz,dimBlock>>>(g,Mx+2,My+2,Mz+1,bz1,bzm,jx,jy);
}
void Initialize(double *Gg_b, double *G1, double *mask){
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	initialize<<<n_blocks,block_size>>>(Gg_b, G1, mask, N_g, M);
}
void Side(double *phi_side, double *phi) {
	Zero(phi_side,M); SetBoundaries(phi);
	Add(phi_side+jx,phi,M-jx); Add(phi_side,phi+jx,M-jx);
	Add(phi_side+jy,phi,M-jy); Add(phi_side,phi+jy,M-jy);
	Add(phi_side+1, phi,M-1);  Add(phi_side,phi+1, M-1);
}
void Propagate(double *G, double *G1, int s_from, int s_to) {
	double *gs = G+M*(s_to-1), *gs_1 = G+M*(s_from-1), *g = G1;
	SetBoundaries(gs_1);
	Times(gs+jx,gs_1,g+jx,M-jx); AddTimes(gs,gs_1+jx,g,M-jx);
	AddTimes(gs+jy,gs_1,g+jy,M-jy); AddTimes(gs,gs_1+jy,g,M-jy);
    AddTimes(gs+1, gs_1,g+1, M-1);  AddTimes(gs,gs_1+1, g,M-1);
    Norm(gs,1.0/6.0,M);
};
void ComputePhi(double *phi, double *G1, double *u, double *Gg_f, double *Gg_b, int sol, int pol){
	Boltzmann(phi+sol,u+sol); Boltzmann(G1,u+pol); SetBoundaries(G1);
	Times(Gg_f,G1,mask,M); 
	for (s=2; s<=N_g/2; s++) Propagate(Gg_f,G1,s-1,s); 

	s = N_g/2;  Zero(phi+pol,M);   
	if (N_g%2 == 1) {  
		Cp(Gg_b+((s+1)%2)*M,Gg_f+(s-1)*M,M);
		Propagate(Gg_b,G1,((s+1)%2)+1,(s%2)+1); 
		AddTimes(phi+pol,Gg_f+(N_g-s)*M,Gg_b+(s%2)*M,M);
		Norm(phi+pol,0.5,M);
	}  else Cp(Gg_b+(s%2)*M,Gg_f+(s-1)*M,M);
	for (s=(N_g+3)/2; s<=N_g; s++) {
		Propagate(Gg_b,G1,((s-1)%2)+1,((s)%2)+1);
		AddTimes(phi+pol,Gg_f+(N_g-s)*M,Gg_b+(s%2)*M,M);
	} 
	GN=Dot(Gg_b+(N_g%2)*M,mask,M); 
	Dubble(phi+pol,G1,2.0*theta/N_g/GN); 
}
void AddG(double *g,double *phitot, double *alpha){int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	addg<<<n_blocks,block_size>>>(g,phitot,alpha,M);
}
void ComputeG(double *g,double *phi, double *u, double *phitot, double *phi_side, double *alpha, int sol, int pol){
	Cp(phitot,phi+sol,M);  Add(phitot,phi+pol,M); 
	Side(phi_side,phi+pol); PutAlpha(g,u+sol,phitot,phi_side,chi,0);
	Side(phi_side,phi+sol); PutAlpha(g+M,u+pol,phitot,phi_side,chi,1.0);
	Cp(alpha,g,M); Add(alpha,g+M,M);
	AddG(g,phitot,alpha); RemoveBoundaries(g); RemoveBoundaries(g+M);
}
void YplusisCtimesX(double *Y, double *X, double C, int M) {
	int n_blocks=(M)/block_size + ((M)%block_size == 0 ? 0:1);
	yplusisctimesx<<<n_blocks,block_size>>>(Y,X,C,M);
}
void Ax(double* A, double* X, int N){//From Ax_B; below B is not used: it is assumed to contain a row of unities.
	double *U = new double[N*N];
	double *S = new double[N];
	double *VT = new double[N*N];
	integer MM = (integer)N, NN = (integer)N;
	integer LDA=MM, LDU=MM, LDVT=NN, INFO, LWORK;
	int lwork;
	double WKOPT;
	double* WORK;
	char JOBU='S'; //'S' is nodig om alleen de eerste N colommen in U te schrijven.
	char JOBVT='A';

	LWORK = -1; //grootte hulpgeheugen aanvragen
	dgesvd_( &JOBU, &JOBVT, &MM, &NN, A, &LDA, S, U, &LDU, VT, &LDVT, &WKOPT, &LWORK, &INFO );
	lwork = (int)WKOPT;
	WORK = (double*)malloc( lwork*sizeof(double) );
	LWORK = (integer)lwork; //nu uitrekenen.
	dgesvd_( &JOBU, &JOBVT, &MM, &NN, A, &LDA, S, U, &LDU, VT, &LDVT, WORK, &LWORK, &INFO );
	if (INFO >0) { //error message genereren
	};
	delete WORK;
	for (int i=0; i<N; i++) X[i]=0;
	for (int i=0; i<N; i++) for (int j=0; j<N; j++) X[i] += U[i*N + j];//*B[j];
	for (int i=0; i<N; i++) {S[i] = X[i]/S[i]; X[i]=0;} //S is use decause it is no longer needed.
	for (int i=0; i<N; i++) for (int j=0; j<N; j++) X[i] += VT[i*N + j]*S[j];
	delete U,S,VT;
}
void DIIS(double *x, double *x_x0, double *xR, double *Aij, double *Apij,double *Ci, int k, int m, int iv) {
	double normC=0; int posi;  
	if (k_diis>m) { k_diis =m;
		for (int i=1; i<m; i++) for (int j=1; j<m; j++) 
		Aij[m*(i-1)+j-1]=Aij[m*i+j]; //remove oldest elements
	}
	for (int i=0; i<k_diis; i++) {posi = k-k_diis+1+i; if (posi<0) posi +=m; 
		Aij[i+m*(k_diis-1)] = Aij[k_diis-1+m*i] = Dot(x_x0+posi*iv, x_x0+k*iv,iv);	}
		// write to (compressed) matrix Apij
	for (int i=0; i<k_diis; i++) for (int j=0; j<k_diis; j++) {
		Apij[j+k_diis*i] = Aij[j+m*i];
	}
	Ax(Apij,Ci,k_diis);		
	for (int i=0; i<k_diis; i++) normC +=Ci[i];
	for (int i=0; i<k_diis; i++) {Ci[i] =Ci[i]/normC; }
	Zero(x,iv);
	posi = k-k_diis+1; if (posi<0) posi +=m; 
		
	YplusisCtimesX(x,xR+posi*iv,Ci[0],iv); //pv = Ci[0]*xR[0];
	for (int i=1; i<k_diis; i++) { 
		posi = k-k_diis+1+i; if (posi<0) posi +=m; 
		YplusisCtimesX(x,xR+posi*iv,Ci[i],iv); 
	}
}
double F() {
	TransferDataToDevice(M, H_mask, mask); 
	double Free_energy=0;
	Zero(x,iv); Zero(x0,iv);
	it=0; j=1; k_diis=1; k=0;
	ComputePhi(phi,G1,u,Gg_f,Gg_b,sol,pol);
	ComputeG(g,phi,u,phitot,phi_side,alpha,sol,pol);
	YplusisCtimesX(x,g,-eta,iv);
	YisAminB(x_x0,x,x0,iv);
	Cp(xR,x,iv); 
	error = sqrt(Dot(g,g,iv));
	printf("DIIS has been notified\n");
	printf("Your guess = %1e \n",error);
	while (error > tolerance && it < 1000) {
		it++;
		Cp(x0,x,iv); 
		ComputePhi(phi,G1,u,Gg_f,Gg_b,sol,pol);
		ComputeG(g,phi,u,phitot,phi_side,alpha,sol,pol);
		k=it % m; k_diis++; //plek voor laatste opslag
		YplusisCtimesX(x,g,-eta,iv);
		Cp(xR+k*iv,x,iv); YisAminB(x_x0+k*iv,x,x0,iv); 	
		DIIS(x,x_x0,xR,Aij,Apij,Ci,k,m,iv); 
		if (it % j == 0) { error = sqrt(Dot(g,g,iv));
			printf("it = %i error = %1e \n",it,error);
			//j = log(error/tolerance); if (j<1) j=1;
		}
	}
       //still need to compute free energy. 
	return Free_energy;
}

main() {  
	hipDeviceReset();

	stat = hipblasCreate(&handle); if (stat !=HIPBLAS_STATUS_SUCCESS) {printf("CUBLAS failed \n");}
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	
	printf("Polymer brush on x-y plane (3-gradient). \n");
	printf("Chain length N (10): "); scanf("%d", &N_g); 
//N_g=10;
//	printf("Grafting density sigma (0.05): "); scanf("%lf", &sigma);
sigma=0.05;
	
	printf("n_layers x (10): "); scanf("%d", &Mx);
	printf("n_layers y (10): "); scanf("%d", &My);
	printf("n_layers z (10): "); scanf("%d", &Mz);
//Mx=10; My=10; Mz = 10;
	theta = sigma*N_g*Mx*My; //here homogeneous grafting on x-y plane assumed (z=1)
//	printf("chi: "); scanf("%lf", &chi);
chi=0;
//	printf("Tolerance (1e-7): "); scanf("%lf", &tolerance);
tolerance = 1e-7;
   	printf("Regularisation parameter, eta (0.1): "); scanf("%lf", &eta);
//eta=0.1;
	printf("Memory depth (m) : "); scanf("%d", &m);
//m=10;
	jx = (My+2)*(Mz+2); jy = Mz+2; M=(Mx+2)*(My+2)*(Mz+2);
	bx1=Mx;   bxm=1; //periodic
	by1=My;   bym=1; //periodic
	bz1=Mz;   bzm=1; // periodic
	pol2 = 2*M; pol = M; sol = 0*M;
	iv=2*M;
	int Ndiv2=N_g/2;

	Aij = new double[m*m]; for (int i=0; i<m*m; i++) Aij[i]=0;
	Ci = new double[m]; for (int i=0; i<m; i++) Ci[i]=0;
	Apij = new double[m*m]; for (int i=0; i<m*m; i++) Apij[i]=0;

	H_phi = new double[M];
	H_mask = new double[M];
	mask = (double*)AllOnDev(M); 
	phi = (double*)AllOnDev(2*M);
	phitot = (double*)AllOnDev(M);
	G1 = (double*)AllOnDev(M); 
	alpha = (double*)AllOnDev(M);
	Gg_f = (double*)AllOnDev(M*Ndiv2); 
	Gg_b = (double*)AllOnDev(M*2);
	phi_side = (double*)AllOnDev(M);  
	x = (double*)AllOnDev(iv); //U_s + U 
	x0 = (double*)AllOnDev(iv);
	g = (double*)AllOnDev(iv);
	xR = (double*)AllOnDev(m*iv);
	x_x0 = (double*)AllOnDev(m*iv);
	DotResult = (double*)AllOnDev(1);

	u = x;
	for (int i=0; i<M; i++) H_mask[i]=0;
	for (int xx=1; xx<=Mx; xx++) for (int yy=1; yy<=My; yy++) H_mask[jx*xx+jy*yy+1]=1.0;
	double Free_energy= F();


//-------------
	//put here your MC code

	

//------------------------------
	TransferDataToHost(H_phi,phi+pol);
	printf("z \t phi_g[z]  \n"); for (int zz=1; zz<=Mz; zz++) printf(" %i \t %1f \n", zz, H_phi[jx+jy+zz]);

	//TransferDataToHost(H_phi,phi+sol);
	//printf("z \t phi_g[z]  \n"); for (int zz=1; zz<=Mz; zz++) printf(" %i \t %1f \n", zz, H_phi[jx+jy+zz]);

	free(H_phi);
	hipFree(phi); hipFree(x);
	hipFree(G1);hipFree(alpha);hipFree(Gg_f);hipFree(Gg_b);hipFree(phi_side);
	hipFree(x_x0);
	hipblasDestroy(handle);
	return(0);
};
